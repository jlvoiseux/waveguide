#include "hip/hip_runtime.h"
#include "wgABC.h"
#include "wgAllocator.h"
#include "wgCommon.h"
#include "wgGrid.h"
#include "wgOutput.h"
#include "wgPlot.h"
#include "wgSource.h"
#include "wgTimer.h"
#include "wgUpdateE.h"
#include "wgUpdateH.h"

#include <cstdio>

int main()
{
	wgTimer timer;
	wgTimerStart(&timer, WG_SIM_STEPS);

	wgGrid grid = {nullptr};

	wgAllocateGrid(&grid, WG_GRID_SIZE_X, WG_GRID_SIZE_Y, WG_GRID_SIZE_Z);
	grid.cdtds = 1.0 / sqrt(3.0);

	wgInitializeECoefficients(&grid);
	wgInitializeHCoefficients(&grid);

	wgInitSource(grid.cdtds, 15.0);
	wgABCInit(&grid);
	wgPlotInit("dipole_sim", WG_PLOT_ALL);
	wgOutputInit("dipole_sim");

	for (int t = 0; t < WG_SIM_STEPS; t++)
	{
		wgTimerFrameStart(&timer);

		wgUpdateH(&grid);
		wgUpdateE(&grid);

		int centerX = (WG_GRID_SIZE_X - 1) / 2;
		int centerY = WG_GRID_SIZE_Y / 2;
		int centerZ = WG_GRID_SIZE_Z / 2;

		double sourceValue = wgGetSourceValue((double)t, 0.0);
		grid.pEx[WG_IDX3(centerX, centerY, centerZ, WG_GRID_SIZE_Y, WG_GRID_SIZE_Z)] += sourceValue;

		if (t == WG_SIM_STEPS - 1)
		{
			if (abs(grid.pEx[WG_IDX3(centerX, centerY, centerZ, WG_GRID_SIZE_Y, WG_GRID_SIZE_Z)] - 0.00030464571644239882) > 0.00000001)
			{
				printf("ERROR\n");
			}
		}

		wgABCApply(&grid);

		wgPlotFrame(&grid, t);

		wgTimerFrameEnd(&timer);
	}

	wgOutputCleanup();
	wgPlotCleanup();
	wgABCCleanup();
	wgFreeGrid(&grid);

	printf("Total simulation time: %.3f seconds\n", wgTimerGetTotalTime(&timer));
	printf("Median frame time: %.3f ms\n", wgTimerGetMedianFrameTime(&timer) * 1000.0);

	wgTimerCleanup(&timer);

	return 0;
}