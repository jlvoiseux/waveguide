#include "hip/hip_runtime.h"
#include "wgABC.h"
#include "wgAllocator.h"
#include "wgCommon.h"
#include "wgGrid.h"
#include "wgOutput.h"
#include "wgPlot.h"
#include "wgSource.h"
#include "wgUpdateE.h"
#include "wgUpdateH.h"

int main()
{
	wgGrid grid = {nullptr};

	wgAllocateGrid(&grid, WG_GRID_SIZE, WG_GRID_SIZE, WG_GRID_SIZE);
	grid.cdtds = 1.0 / sqrt(3.0);

	wgInitializeECoefficients(&grid);
	wgInitializeHCoefficients(&grid);

	wgInitSource(grid.cdtds, 15.0);
	wgABCInit(&grid);
	wgPlotInit("dipole_sim");
	wgOutputInit("dipole_sim");

	for (int t = 0; t < WG_SIM_STEPS; t++)
	{
		wgUpdateH(&grid);
		wgUpdateE(&grid);

		int centerX = (WG_GRID_SIZE - 1) / 2;
		int centerY = WG_GRID_SIZE / 2;
		int centerZ = WG_GRID_SIZE / 2;
		grid.pEx[WG_IDX3(centerX, centerY, centerZ, WG_GRID_SIZE, WG_GRID_SIZE)] += wgGetSourceValue((double)t, 0.0);

		wgABCApply(&grid);

		if (t % 5 == 0)
		{
			wgPlotFrame(&grid, t);
			wgOutputFrame(&grid, t);
		}
	}

	wgOutputCleanup();
	wgPlotCleanup();
	wgABCCleanup();
	wgFreeGrid(&grid);

	return 0;
}