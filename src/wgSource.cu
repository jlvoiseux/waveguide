#include "hip/hip_runtime.h"
#include "wgCommon.h"
#include "wgSource.h"

#include <cstdio>

static double gCdtds			   = 0.0;
static double gPointsPerWavelength = 0.0;

void wgInitSource(double cdtds, double pointsPerWavelength)
{
	if (pointsPerWavelength <= 0.0)
	{
		fprintf(stderr, "Points per wavelength must be positive\n");
		exit(1);
	}

	gCdtds				 = cdtds;
	gPointsPerWavelength = pointsPerWavelength;
}

double wgGetSourceValue(double time, double location)
{
	if (gPointsPerWavelength <= 0.0)
	{
		fprintf(stderr, "Source not initialized. Call wgInitSource first\n");
		exit(1);
	}

	double arg = WG_PI * ((gCdtds * time - location) / gPointsPerWavelength - 1.0);
	arg		   = arg * arg;

	return (1.0 - 2.0 * arg) * exp(-arg);
}