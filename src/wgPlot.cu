#include "hip/hip_runtime.h"
// wgPlot.cu
#include "wgCommon.h"
#include "wgPlot.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef _WIN32
#define POPEN _popen
#define PCLOSE _pclose
#else
#define POPEN popen
#define PCLOSE pclose
#endif

static FILE*	  gPlotPipeX  = NULL;
static FILE*	  gPlotPipeY  = NULL;
static FILE*	  gPlotPipe3D = NULL;
static char		  gBaseFileName[256];
static wgPlotMode gPlotMode;

static double scaleValue(double val)
{
	double magnitude = fabs(val / 0.3);
	return magnitude > 0.0 ? log10(magnitude) : -3.0;
}

static void initSlicePlots(void)
{
	const char* commonSettings = "set term gif animate delay 1\n"
								 "set size square\n"
								 "set palette defined (-3 'blue', -2 'cyan', -1 'yellow', 0 'red')\n"
								 "set cbrange [-3:0]\n"
								 "set pm3d map corners2color c4\n"
								 "set pm3d interpolate 0,0\n"
								 "unset key\n"
								 "set xrange [0:31]\n"
								 "set yrange [0:31]\n";

	gPlotPipeX = POPEN("\"" GNUPLOT_EXECUTABLE "\" --persist", "w");
	gPlotPipeY = POPEN("\"" GNUPLOT_EXECUTABLE "\" --persist", "w");

	if (!gPlotPipeX || !gPlotPipeY)
	{
		fprintf(stderr, "Failed to open gnuplot pipe for slices\n");
		exit(1);
	}

	char fileNameX[512], fileNameY[512];
	sprintf(fileNameX, "%s-x.gif", gBaseFileName);
	sprintf(fileNameY, "%s-y.gif", gBaseFileName);

	fprintf(gPlotPipeX, "%s", commonSettings);
	fprintf(gPlotPipeX, "set output '%s'\n", fileNameX);
	fprintf(gPlotPipeX, "set xlabel 'Y'\n");
	fprintf(gPlotPipeX, "set ylabel 'Z'\n");

	fprintf(gPlotPipeY, "%s", commonSettings);
	fprintf(gPlotPipeY, "set output '%s'\n", fileNameY);
	fprintf(gPlotPipeY, "set xlabel 'X'\n");
	fprintf(gPlotPipeY, "set ylabel 'Z'\n");
}

static void init3DPlot(void)
{
	char fileName[512];
	sprintf(fileName, "%s-3d.gif", gBaseFileName);

	gPlotPipe3D = POPEN("\"" GNUPLOT_EXECUTABLE "\" --persist", "w");
	if (!gPlotPipe3D)
	{
		fprintf(stderr, "Failed to open gnuplot pipe for 3D\n");
		exit(1);
	}

	fprintf(gPlotPipe3D, "set term gif animate delay 10\n");
	fprintf(gPlotPipe3D, "set output '%s'\n", fileName);
	fprintf(gPlotPipe3D, "set xlabel 'X'\n");
	fprintf(gPlotPipe3D, "set ylabel 'Y'\n");
	fprintf(gPlotPipe3D, "set zlabel 'Z'\n");
	fprintf(gPlotPipe3D, "set view 60,30\n");
	fprintf(gPlotPipe3D, "set palette defined (-3 'blue', -2 'cyan', -1 'yellow', 0 'red')\n");
	fprintf(gPlotPipe3D, "set cbrange [-3:0]\n");
	fprintf(gPlotPipe3D, "set pm3d\n");
	fprintf(gPlotPipe3D, "set hidden3d\n");
}

void wgPlotInit(const char* pBaseFileName, wgPlotMode mode)
{
	if (!pBaseFileName)
	{
		fprintf(stderr, "Invalid base filename\n");
		exit(1);
	}

#ifdef GNUPLOT_EXECUTABLE
	sprintf(gBaseFileName, "%s", pBaseFileName);
	gPlotMode = mode;

	if (mode & WG_PLOT_SLICES)
		initSlicePlots();
	if (mode & WG_PLOT_3D)
		init3DPlot();
#else
	fprintf(stderr, "Gnuplot not available\n");
	exit(1);
#endif
}

static void plotSlices(wgGrid* pGrid, int frame)
{
	int centerX = (pGrid->sizeX - 1) / 2;
	int centerY = pGrid->sizeY / 2;

	// X slice
	fprintf(gPlotPipeX, "set title 'X Slice (Frame %d)'\n", frame);
	fprintf(gPlotPipeX, "splot '-' using 1:2:3 with pm3d\n");

	for (int y = 0; y < pGrid->sizeY; y++)
	{
		for (int z = 0; z < pGrid->sizeZ; z++)
		{
			double ex = pGrid->pEx[WG_IDX3(centerX, y, z, pGrid->sizeY, pGrid->sizeZ)];
			fprintf(gPlotPipeX, "%d %d %f\n", y, z, scaleValue(ex));
		}
		fprintf(gPlotPipeX, "\n");
	}
	fprintf(gPlotPipeX, "e\n");
	fflush(gPlotPipeX);

	// Y slice
	fprintf(gPlotPipeY, "set title 'Y Slice (Frame %d)'\n", frame);
	fprintf(gPlotPipeY, "splot '-' using 1:2:3 with pm3d\n");

	for (int x = 0; x < pGrid->sizeX - 1; x++)
	{
		for (int z = 0; z < pGrid->sizeZ; z++)
		{
			double ex = pGrid->pEx[WG_IDX3(x, centerY, z, pGrid->sizeY, pGrid->sizeZ)];
			fprintf(gPlotPipeY, "%d %d %f\n", x, z, scaleValue(ex));
		}
		fprintf(gPlotPipeY, "\n");
	}
	fprintf(gPlotPipeY, "e\n");
	fflush(gPlotPipeY);
}

static void plot3D(wgGrid* pGrid, int frame)
{
	fprintf(gPlotPipe3D, "splot '-' using 1:2:3:4 with pm3d title 'Frame %d'\n", frame);

	for (int x = 0; x < pGrid->sizeX - 1; x++)
	{
		for (int y = 0; y < pGrid->sizeY - 1; y++)
		{
			for (int z = 0; z < pGrid->sizeZ - 1; z++)
			{
				double ex		 = pGrid->pEx[WG_IDX3(x, y, z, pGrid->sizeY, pGrid->sizeZ)];
				double ey		 = pGrid->pEy[WG_IDX3(x, y, z, pGrid->sizeY - 1, pGrid->sizeZ)];
				double ez		 = pGrid->pEz[WG_IDX3(x, y, z, pGrid->sizeY, pGrid->sizeZ - 1)];
				double magnitude = sqrt(ex * ex + ey * ey + ez * ez);

				fprintf(gPlotPipe3D, "%d %d %d %f\n", x, y, z, scaleValue(magnitude));
			}
			fprintf(gPlotPipe3D, "\n");
		}
		fprintf(gPlotPipe3D, "\n");
	}

	fprintf(gPlotPipe3D, "e\n");
	fflush(gPlotPipe3D);
}

void wgPlotFrame(wgGrid* pGrid, int frame)
{
#ifdef GNUPLOT_EXECUTABLE
	if (!pGrid || gPlotMode & WG_PLOT_NONE)
		return;

	if (gPlotMode & WG_PLOT_SLICES)
		plotSlices(pGrid, frame);
	if (gPlotMode & WG_PLOT_3D)
		plot3D(pGrid, frame);
#endif
}

void wgPlotCleanup(void)
{
#ifdef GNUPLOT_EXECUTABLE
	if (gPlotPipeX)
	{
		PCLOSE(gPlotPipeX);
		gPlotPipeX = NULL;
	}
	if (gPlotPipeY)
	{
		PCLOSE(gPlotPipeY);
		gPlotPipeY = NULL;
	}
	if (gPlotPipe3D)
	{
		PCLOSE(gPlotPipe3D);
		gPlotPipe3D = NULL;
	}
#endif
}