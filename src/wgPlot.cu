#include "hip/hip_runtime.h"
#include "wgCommon.h"
#include "wgPlot.h"

#include <cstdio>
#include <cstdlib>

#ifdef _WIN32
#define POPEN _popen
#define PCLOSE _pclose
#else
#define POPEN popen
#define PCLOSE pclose
#endif

static FILE* gPlotPipe = nullptr;

void wgPlotInit(const char* baseFileName)
{
	if (!baseFileName)
	{
		fprintf(stderr, "Invalid base filename\n");
		exit(1);
	}

#ifdef GNUPLOT_EXECUTABLE
	gPlotPipe = POPEN("\"" GNUPLOT_EXECUTABLE "\" --persist", "w");
	if (!gPlotPipe)
	{
		fprintf(stderr, "Failed to open gnuplot pipe\n");
		exit(1);
	}

	fprintf(gPlotPipe, "set term gif animate delay 10\n");
	fprintf(gPlotPipe, "set output '%s.gif'\n", baseFileName);
	fprintf(gPlotPipe, "set xlabel 'X'\n");
	fprintf(gPlotPipe, "set ylabel 'Y'\n");
	fprintf(gPlotPipe, "set zlabel 'Z'\n");
	fprintf(gPlotPipe, "set view 60,30\n");
	fprintf(gPlotPipe, "set palette defined (-0.01 'blue', 0 'white', 0.01 'red')\n");
	fprintf(gPlotPipe, "set cbrange [-0.01:0.01]\n");
	fprintf(gPlotPipe, "set pm3d\n");
	fprintf(gPlotPipe, "set hidden3d\n");
#else
	fprintf(stderr, "Gnuplot not available\n");
	exit(1);
#endif
}

void wgPlotFrame(wgGrid* pGrid, int frame)
{
#ifdef GNUPLOT_EXECUTABLE
	if (!gPlotPipe || !pGrid)
		return;

	fprintf(gPlotPipe, "splot '-' using 1:2:3:4 with pm3d title 'Frame %d'\n", frame);

	for (int x = 0; x < pGrid->sizeX - 1; x++)
	{
		for (int y = 0; y < pGrid->sizeY - 1; y++)
		{
			for (int z = 0; z < pGrid->sizeZ - 1; z++)
			{
				double ex		 = pGrid->pEx[WG_IDX3(x, y, z, pGrid->sizeY, pGrid->sizeZ)];
				double ey		 = pGrid->pEy[WG_IDX3(x, y, z, pGrid->sizeY - 1, pGrid->sizeZ)];
				double ez		 = pGrid->pEz[WG_IDX3(x, y, z, pGrid->sizeY, pGrid->sizeZ - 1)];
				double magnitude = sqrt(ex * ex + ey * ey + ez * ez);

				magnitude = magnitude > 1.0 ? 1.0 : (magnitude < -1.0 ? -1.0 : magnitude);

				fprintf(gPlotPipe, "%d %d %d %f\n", x, y, z, magnitude);
			}
			fprintf(gPlotPipe, "\n");
		}
		fprintf(gPlotPipe, "\n");
	}

	fprintf(gPlotPipe, "e\n");
	fflush(gPlotPipe);
#endif
}

void wgPlotCleanup()
{
#ifdef GNUPLOT_EXECUTABLE
	if (gPlotPipe)
	{
		PCLOSE(gPlotPipe);
		gPlotPipe = nullptr;
	}
#endif
}